
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16
#define RANDOM_MN_RANGE 64

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
struct Matrix {
    int width;
    int height;
    int stride; 
    float* elements;
};

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to mak sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}


extern "C" {
    void MatMul( Matrix *A, Matrix *B, Matrix *C ){

        Matrix d_A, d_B, d_C;

        // Matrix d_A
        d_A.width    =   A->width;
        d_A.stride   =   A->width;
        d_A.height   =   A->height;
        size_t sizeA =   A->width * A->height * sizeof(float);
        // dynamically allocate cudaMemory for elemenst array
        hipMalloc(&d_A.elements, sizeA);
        hipMemcpy(d_A.elements, A->elements, sizeA, hipMemcpyHostToDevice);

        // Matrix d_B
        d_B.width    =   B->width;
        d_B.stride   =   B->width;
        d_B.height   =   B->height;
        size_t sizeB =   B->width * B->height * sizeof(float);
        // dynamically allocate cudaMemory for elemenst array
        hipMalloc(&d_B.elements, sizeB);
        hipMemcpy(d_B.elements, B->elements, sizeB, hipMemcpyHostToDevice);

        // Matrix d_C
        d_C.width    =   C->width;
        d_C.stride   =   C->width;
        d_C.height   =   C->height;
        size_t sizeC =   C->width * C->height * sizeof(float);

        // dynamically allocate cudaMemory for elemenst array
        hipMalloc(&d_C.elements, sizeC);

        // 16 * 16 = 256 threads per block
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

        // Blocks per grid
        dim3 dimGrid(B->width / dimBlock.x, A->height / dimBlock.y);

        // calling the Kernel
        MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

        // copy results from result matrix C to the host again
        hipMemcpy(C->elements, d_C.elements, sizeC, hipMemcpyDeviceToHost);

        printf("A is %f\n", A->elements[0]);
        printf("B is %f\n", B->elements[0]);
        printf("C is %f\n", C->elements[0]);


        // free the cuda memory
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
    }
}


